#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// downsweep exclusive scan kernel function
__global__ void downsweep_kernel(int two_dplus1, int two_d, int* result, int bound) {
  int global_i = (blockIdx.x * blockDim.x + threadIdx.x);
  //int t = result[global_i + two_d - 1];
  if (global_i < bound){
    int t = result[global_i * two_dplus1 + two_d - 1];
    result[global_i * two_dplus1 + two_d - 1] = result[global_i * two_dplus1 + two_dplus1 - 1];
    result[global_i * two_dplus1 + two_dplus1 - 1] += t;
  }
}

/*
Input: [1, 2, 2, 1, 1, 1, 3, 5, 3, 3]
Bitmask: [0, 0, 0, 0, 0, 0, 0, 0, 0, 0] // cuda malloc 0's and update from there
Given index i in bitmask, =1 if input[i] = input[i+1]
Wanted output: [0, 1, 0, 1, 1, 0, 0, 0, 1, 0]
Ex_scan: 

   */

__global__ void mid_kernel(int last, int*result){
  result[last] = 0;
}

// upsweep exclusive scan kernel function
// TODO: numBlocks = blockDim.x?
__global__ void upsweep_kernel(int two_dplus1, int two_d, int* result, int bound) {
  int global_i = (blockIdx.x * blockDim.x  + threadIdx.x) ;
  if (global_i < bound){
    result[global_i*two_dplus1 + two_dplus1 - 1] += result[global_i*two_dplus1 + two_d - 1];
  }
}

__global__ void find_flags(int* input, int bound, int* result) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index < bound) {
    if (input[index] == input[index + 1]) {
      result[index] = 1;
    } else {
      result[index] = 0;
    }
  //} else {
    //result[index] = 0;
  }
}


// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
// TODO: assums that result starts with input already copied over
void exclusive_scan(int* input, int N, int* result)
{
  // upsweep
  N = nextPow2(N);
  for (int two_d = 1; two_d <= N/2; two_d*=2) {
    int two_dplus1 = 2*two_d;
    int total_threads_per_it = N / two_dplus1;
    int num_blocks = int(total_threads_per_it / THREADS_PER_BLOCK)+1;
    upsweep_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, result, total_threads_per_it);
  }

  //result[N - 1] = 0; Can I do this or do I need a kernel?
  mid_kernel<<< 1,1 >>>(N-1,result);

  // downsweep
  for (int two_d = N / 2; two_d >= 1; two_d /= 2) {
    int two_dplus1 = 2 * two_d;
    int total_threads_per_it = N / two_dplus1;
    int num_blocks = int(total_threads_per_it / THREADS_PER_BLOCK) + 1;
    downsweep_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, result, total_threads_per_it);

  }

  }



//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void find_output(int* input, int bound, int* output) {
  int index = threadIdx.x + (blockDim.x * blockIdx.x);
  if (index < bound){
    if (input[index] == 1 && index == 0) {
      output[0] = 0;
    }else if (input[index] != input[index+1]) {
      output[input[index]] = index;
    }
  }
}
// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
// TODO: What to do with device_output?
int find_repeats(int* device_input, int length, int* device_output) {
    // int* flags = new int[length];
    // int* flags_scanned = new int[length];
    int block_amt = int(length/ THREADS_PER_BLOCK) + 1;
    find_flags<<<block_amt,THREADS_PER_BLOCK>>>(device_input, length-1, device_output);
    exclusive_scan(device_input,length,device_output);
    int output_size;
    hipMemcpy(&output_size, &device_output[length-1],sizeof(int), hipMemcpyDeviceToHost);
    int* device_tmp = nullptr;
    hipMalloc(&device_tmp,length * sizeof(int));
    hipMemcpy(device_tmp,device_output,length*sizeof(int), hipMemcpyDeviceToDevice);
    find_output<<<block_amt,THREADS_PER_BLOCK>>>(device_tmp, length-1, device_output);
    hipFree(device_tmp);
    return output_size;

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
